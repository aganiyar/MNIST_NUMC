#include "hip/hip_runtime.h"
// layers
#include <layers/layer.cuh>
#include <layers/reluLayer.cuh>

// numC
#include <numC/npGPUArray.cuh>
#include <numC/gpuConfig.cuh>
#include <numC/npFunctions.cuh>
#include <numC/customKernels.cuh>

// relu layer definiton

// default constructor
ReLULayer::ReLULayer()
{
    ;
}

// copy constructor
ReLULayer::ReLULayer(const ReLULayer &L)
{
    this->cache = L.cache;
}

// assignment operator
void ReLULayer::operator=(const ReLULayer &L)
{
    this->cache = L.cache;
}

// ################################# forward pass ##############################################
np::ArrayGPU<float> ReLULayer::forward(const np::ArrayGPU<float> &X, const std::string &mode)
{
    /* Computes the forward pass for a layer of rectified linear units (ReLUs).

        Input:
        - X: Inputs, of any shape

        Returns:
        - out: Output, of the same shape as x

        Also stores:
        - cache: x, for backpropagation
    */
    auto out = np::maximum<float>(X, 0);
    if (mode == "train")
        this->cache = X;
    return out;
}
// #############################################################################################

// ################################# backward pass #############################################
np::ArrayGPU<float> ReLULayer::backward(const np::ArrayGPU<float> &dout)
{
    /* Computes the backward pass for a layer of rectified linear units (ReLUs).

        Input:
        - dout: Upstream derivatives, of any shape
        - cache: Input x, of same shape as dout

        Returns:
        - dx: Gradient with respect to x
    */
    np::ArrayGPU<float> dX(dout.rows(), dout.cols());
    int size = dout.size();
    const int BLOCK_SIZE = np::GPU_NUM_CUDA_CORE;
    dim3 block(BLOCK_SIZE);
    dim3 grid(ceil(std::min<int>(static_cast<float>(size)/block.x, 2 * np::GPU_NUM_SM)));

    kernelReLUBackward<float><<<grid, block>>>(dout.mat, cache.mat, dX.mat, size);
    hipDeviceSynchronize();

    return dX;
    // #############################################################################################
}